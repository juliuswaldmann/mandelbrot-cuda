#include <stdint.h>
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <memory.h>

#include <string>

#include <iostream>

#include <atomic>

#include <mutex>
#include <vector>

#define maxiterations 100


//Cuda functions
__device__ int8_t get_iterations(double a1, double b1){
    int8_t iterations = 0;
    double a = a1;
    double b = b1;
    while(a*a + b*b < 4 && iterations < maxiterations){
        double a_new = a*a - b*b + a1;
        double b_new = 2*a*b + b1;
        a = a_new;
        b = b_new;
        iterations++;
    }
    return iterations;
}

__device__ ulong3 hueToRGB(int hue) {
    // hue is an integer between 0 and 360
    // this function converts it to an RGB color
    // the output is a 3-element array of integers between 0 and 255
    ulong3 rgb;
    int h = hue;
    if (h < 60) {
        rgb.x = 255;
        rgb.y = h * 255 / 60;
        rgb.z = 0;
    } else if (h < 120) {
        rgb.x = 255 - (h - 60) * 255 / 60;
        rgb.y = 255;
        rgb.z = 0;
    } else if (h < 180) {
        rgb.x = 0;
        rgb.y = 255;
        rgb.z = (h - 120) * 255 / 60;
    } else if (h < 240) {
        rgb.x = 0;
        rgb.y = 255 - (h - 180) * 255 / 60;
        rgb.z = 255;
    } else if (h < 300) {
        rgb.x = (h - 240) * 255 / 60;
        rgb.y = 0;
        rgb.z = 255;
    } else {
        rgb.x = 255;
        rgb.y = 0;
        rgb.z = 255 - (h - 300) * 255 / 60;
    }
    return rgb;
}

__global__ void stuffKernel(ulong3 *arrayPtr, dim3 dimensions, dim3 threadAreaSize)
{
    for(int x = 0; x < threadAreaSize.x; x++){
        for(int y = 0; y < threadAreaSize.y; y++){
            double a = (threadIdx.x + x / threadAreaSize.x) / dimensions.x * 4 - 2;
            double b = (threadIdx.y + y / threadAreaSize.y) / dimensions.y * 4 - 2;
            int8_t iterations = get_iterations(a, b);
            ulong3 color = {0, 0, 0};
            if(iterations < maxiterations){
                color = hueToRGB((iterations % 10) / 10 * 360);
            }

            arrayPtr[(threadIdx.y * threadAreaSize.y + y) * 1080 + threadIdx.x * threadAreaSize.x + x] = color;

        }
    }

}

//Cpu functions
int main()

{
    //set Cuda Device
    hipDeviceReset();
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }
    else {
        fprintf(stdout, "Successfully set CudaDevice\n");
    }

    //cudaStream_t *mainStream;
    //cudaStreamCreate(mainStream);

    

    int imageWidth = 1080;
    int imageHeight = 1080;

    int long3size = 24;
    //long arrayBytes = imageWidth * imageHeight * long3size;

    fprintf(stdout, "hallo ey");

    ulong3 *arrayPtr;
    hipMalloc((void **) &arrayPtr, 8 * sizeof(ulong3));

    fprintf(stdout, "hallo eyoo");

    dim3 threads_per_block = dim3(32, 32, 1);
    dim3 blocks_per_grid = dim3(1, 1, 1);
    dim3 dimensions = dim3(threads_per_block.x * blocks_per_grid.x, threads_per_block.y * blocks_per_grid.y, 1);
    dim3 threadAreaSize = dim3(imageWidth/dimensions.x, imageHeight/dimensions.y, 1);

    fprintf(stdout, "Hallovorvor");

    //uint3 arraySize = {imageWidth, imageHeight, 1};

    //stuffKernel<<<blocks_per_grid, threads_per_block, 0, *mainStream>>>(arrayPtr, dimensions, threadAreaSize);

    


    fprintf(stdout, "Hallovor");

    cudaStatus = hipDeviceSynchronize();

    
    fprintf(stdout, "Hallo");
    
    //cudaStatus = cudaDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    while (true) {
        
    }

    return 0;
}
